#include "hip/hip_runtime.h"
//TODO: separate cu kernel files
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
//#include "cuPrintf.cu"

using namespace std;
using namespace cv;

__global__ void kernCalcBlockHist(
	unsigned char * src,
	int rows,
	int cols,
	int strideX,
	int strideY,
	int blockSizeX,
	int blockSizeY,
	unsigned int *** outHist
)
{
	int idx_x = (strideX * blockIdx.x) + threadIdx.x;
	int idx_y = (strideY * blockIdx.y) + (cols * threadIdx.y);
	//cout<<"("<<idx_x<<","<<idx_y<<")"<<endl;
	int idx = idx_y + idx_x;
	//cuPrintf("%d\n", idx);
	atomicAdd(&(outHist[blockIdx.x][blockIdx.y][src[idx]]), 1);
}

__global__ void cudaCalcCornerBlockHist(
		unsigned char * src, 
		int rows, 
		int cols, 
		int blockSizeX, 
		int blockSizeY, 
		int beginX, 
		int beginY, 
		unsigned int *** outHist
)
{
	int idx = rows * (threadIdx.y + beginY) + (threadIdx.x + beginX);
	atomicAdd(&(outHist[0][0][src[idx]]), 1);
}


void calcCornerBlockHist(const Mat src, int blockSizeX, int blockSizeY, int beginX, int beginY, unsigned char * outHist){
	unsigned char *input = (unsigned char*) src.data;
	int count = 0;
	int bin;
	for (int j = beginY; j < beginY + blockSizeY; j++){
		for (int i = beginX; i < beginX + blockSizeX; i++){
			bin = input[src.rows * j + i];
			//cout<<"index = "<<src.rows * j + i<<endl;
			outHist[bin]++;
			count++;
		}
	}
	cout<<endl;
	cout<<"count = "<<count<<endl;
}

void processHistogram(unsigned char * hist, int max){
	float mean = 0;
	float sum = 0;
	int occurences = 0;
	
	for (int i = 0; i < max; i++){
		cout<<(int)hist[i]<<" ";
		sum += (float)i * hist[i];
		occurences += hist[i];
	}
	mean = sum/occurences;
	cout<<endl;
	cout<<"sum = "<<sum<<endl;
	cout<<"occurences = "<<occurences<<endl;
	cout<<"mean = "<<setprecision(5)<<mean<<endl;
}

void processHistogram(unsigned int * hist, int max){
	float mean = 0;
	float sum = 0;
	int occurences = 0;
	
	for (int i = 0; i < max; i++){
		cout<<(int)hist[i]<<" ";
		sum += (float)i * hist[i];
		occurences += hist[i];
	}
	mean = sum/occurences;
	cout<<endl;
	cout<<"sum = "<<sum<<endl;
	cout<<"occurences = "<<occurences<<endl;
	cout<<"mean = "<<setprecision(5)<<mean<<endl;
}

int main(int argc, char** argv){
	if (argc != 2){
		cout<<"usage:\n";
		cout<<"  "<<argv[0]<<" [image file]\n";
		return -1;
	}
	
	Mat matSource;
	matSource = imread(argv[1], 0);
	
	int blockSizeX = 32;
	int blockSizeY = 32;
	int startX = 0;
	int startY = 0;
	
	unsigned char hist[256] = {0};
	
	//namedWindow("Original", CV_WINDOW_AUTOSIZE);
	//imshow("Original", matSource);
	cout<<"source rows: "<<matSource.rows<<endl;
	cout<<"source cols: "<<matSource.cols<<endl;
	
	// timer
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	/**
	hipEventRecord(start, 0);
	// function call
	calcCornerBlockHist(matSource, blockSizeX, blockSizeY, startX, startY, hist);
	// for 128x128 block size, why total occurences is less than 128x128=16384?
	
	// timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout<<"CPU histogram took "<<setprecision(5)<<elapsedTime*1000<<" ns"<<endl;
	
	cout<<"histogram"<<endl;
	processHistogram(hist, 256);
	cout<<endl;
	*/
	// cuda
	int strideX = blockSizeX/2;
	int strideY = blockSizeY/2;
	dim3 numGrid = dim3(matSource.cols/strideX - 1, matSource.rows/strideY - 1, 1);
	dim3 numBlock = dim3(blockSizeX, blockSizeY, 1);
	
	int totalBlocks = (matSource.cols/strideX) * (matSource.rows/strideY);
	cout<<"total blocks = "<<totalBlocks<<endl;
	cout<<"numGrid = ("<<matSource.cols/strideX<<", "<<matSource.rows/strideY<<")"<<endl;
	cout<<"numBlock = ("<<blockSizeX<<", "<<blockSizeY<<")"<<endl;
		
	unsigned char* host_image = matSource.data;
	unsigned int host_hist[blockSizeX][blockSizeY][256];
	
	unsigned char* dev_image;
	size_t size = matSource.rows*matSource.cols*sizeof(unsigned char);
	hipMalloc(&dev_image, size);
	hipMemcpy(dev_image, host_image, size, hipMemcpyHostToDevice);
	
	unsigned int *** dev_hist;
	hipMalloc(&dev_hist, blockSizeX*blockSizeY*256*sizeof(unsigned int));
	hipMemset(dev_hist, 0, blockSizeX*blockSizeY*256*sizeof(unsigned int));
	
	hipEventRecord(start, 0);
	cudaCalcCornerBlockHist<<<1, numBlock>>>(dev_image, matSource.rows, matSource.cols, blockSizeX, blockSizeY, startX, startY, dev_hist);
	
	//cudaPrintfinit();
	
	//kernCalcBlockHist<<<numGrid, numBlock>>>(dev_image, matSource.rows, matSource.cols, strideX, strideY, blockSizeX, blockSizeY, dev_hist);
	
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout<<"GPU histogram took "<<setprecision(5)<<elapsedTime*1000<<" ns"<<endl;
	
	hipMemcpy(host_hist, dev_hist, blockSizeX*blockSizeY*256*sizeof(unsigned int), hipMemcpyDeviceToHost);
	cout<<"histogram from cuda"<<endl;
	//processHistogram(host_hist, 256);
	processHistogram(host_hist[0][0], 256);
	
	
	
	// cleanup
	hipFree(dev_image);
	hipFree(dev_hist);
	hipDeviceReset();
	return 0;
}
