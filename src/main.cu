#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

__global__ void cudaCalcCornerBlockHist(unsigned char * src, 
										int rows, 
										int cols, 
										int blockSizeX, 
										int blockSizeY, 
										int beginX, 
										int beginY, 
										unsigned int * outHist){
	int idx = rows * (threadIdx.y + beginY) + (threadIdx.x + beginX);
	atomicAdd(&(outHist[src[idx]]), 1);
}


void calcCornerBlockHist(const Mat src, int blockSizeX, int blockSizeY, int beginX, int beginY, unsigned char * outHist){
	unsigned char *input = (unsigned char*) src.data;
	int count = 0;
	int bin;
	for (int j = beginY; j < beginY + blockSizeY; j++){
		for (int i = beginX; i < beginX + blockSizeX; i++){
			bin = input[src.rows * j + i];
			//cout<<"index = "<<src.rows * j + i<<endl;
			outHist[bin]++;
			count++;
		}
	}
	cout<<endl;
	cout<<"count = "<<count<<endl;
}

void processHistogram(unsigned char * hist, int max){
	float mean = 0;
	float sum = 0;
	int occurences = 0;
	
	for (int i = 0; i < max; i++){
		cout<<(int)hist[i]<<" ";
		sum += (float)i * hist[i];
		occurences += hist[i];
	}
	mean = sum/occurences;
	cout<<endl;
	cout<<"sum = "<<sum<<endl;
	cout<<"occurences = "<<occurences<<endl;
	cout<<"mean = "<<setprecision(5)<<mean<<endl;
}

void processHistogram(unsigned int * hist, int max){
	float mean = 0;
	float sum = 0;
	int occurences = 0;
	
	for (int i = 0; i < max; i++){
		cout<<(int)hist[i]<<" ";
		sum += (float)i * hist[i];
		occurences += hist[i];
	}
	mean = sum/occurences;
	cout<<endl;
	cout<<"sum = "<<sum<<endl;
	cout<<"occurences = "<<occurences<<endl;
	cout<<"mean = "<<setprecision(5)<<mean<<endl;
}

int main(int argc, char** argv){
	if (argc != 2){
		cout<<"usage:\n";
		cout<<"  "<<argv[0]<<" [image file]\n";
		return -1;
	}
	
	hipEvent_t start, stop;
	
	Mat matSource;
	matSource = imread(argv[1], 0);
	
	int blockSizeX = 32;
	int blockSizeY = 32;
	int startX = 0;
	int startY = 0;
	
	
	unsigned char hist[256] = {0};
	
	//namedWindow("Original", CV_WINDOW_AUTOSIZE);
	//imshow("Original", matSource);
	cout<<"source rows: "<<matSource.rows<<endl;
	cout<<"source cols: "<<matSource.cols<<endl;
	
	// timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	// function call
	calcCornerBlockHist(matSource, blockSizeX, blockSizeY, startX, startY, hist);
	// for 128x128 block size, why total occurences is less than 128x128=16384?
	
	// timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout<<"CPU histogram took "<<setprecision(5)<<elapsedTime*1000<<" ns"<<endl;
	
	cout<<"histogram"<<endl;
	processHistogram(hist, 256);
	cout<<endl;
	
	// cuda
	unsigned char* host_image = matSource.data;
	unsigned int host_hist[256];
	
	unsigned char* dev_image;
	size_t size = matSource.rows*matSource.cols*sizeof(unsigned char);
	hipMalloc(&dev_image, size);
	hipMemcpy(dev_image, host_image, size, hipMemcpyHostToDevice);
	
	unsigned int * dev_hist;
	hipMalloc(&dev_hist, 256*sizeof(unsigned int));
	hipMemset(dev_hist, 0, 256*sizeof(unsigned int));
	
	hipEventRecord(start, 0);
	cudaCalcCornerBlockHist<<<1, dim3(32,32, 1)>>>(dev_image, matSource.rows, matSource.cols, blockSizeX, blockSizeY, startX, startY, dev_hist);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout<<"GPU histogram took "<<setprecision(5)<<elapsedTime*1000<<" ns"<<endl;
	
	hipMemcpy(host_hist, dev_hist, 256*sizeof(unsigned int), hipMemcpyDeviceToHost);
	cout<<"histogram from cuda"<<endl;
	processHistogram(host_hist, 256);
	
	// cleanup
	hipFree(dev_image);
	hipFree(dev_hist);
	return 0;
}
